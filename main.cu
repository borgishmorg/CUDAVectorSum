#include <hip/hip_runtime.h>

#include <iostream>
#include <fstream>
#include <thread>
#include <chrono>
#include <atomic>

using namespace std;

int n;
int *a = NULL;
int *b = NULL;
int *c = NULL;
atomic<int> cnt(0);
bool run = false;

__global__ void kernel(int* a, int* b, int*c, int l, int r){
    int i = blockIdx.x*blockDim.x+threadIdx.x;
    if(l <= i && i < r)
        c[i] = a[i] + b[i];
}

void slave(int id){
    hipSetDevice(id);
    hipSetDeviceFlags(hipDeviceMapHost);

    cnt++;

    while(!run) this_thread::sleep_for(chrono::milliseconds(20));

    kernel<<<(n/3+31), 32>>>(a, b, c, n/3*id, n/3*(id+1));
    hipDeviceSynchronize();
    cnt++;
}

int main(){
    ifstream in("input.txt");
    ofstream out("output.txt");

    thread slave0(slave, 0);
    thread slave1(slave, 1);

    while(cnt != 2) this_thread::sleep_for(chrono::milliseconds(20));
    
    in >> n;
    
    hipHostAlloc(&a, n*sizeof(int), hipHostMallocMapped);
    hipHostAlloc(&b, n*sizeof(int), hipHostMallocMapped);
    hipHostAlloc(&c, n*sizeof(int), hipHostMallocMapped);

    for(int i = 0; i < n ; i++) in >> a[i];
    for(int i = 0; i < n ; i++) in >> b[i];

    run = true;

    for(int i = n/3*2; i < n; i++)
        c[i] = a[i] + b[i];

    if(slave0.joinable())
        slave0.join();
    if(slave1.joinable())
        slave1.join();

    for(int i = 0; i < n; i++)
        out << c[i] << ' ';
    return 0;
}